#include "hip/hip_runtime.h"

#include "constants.hpp"

#include <iostream>
#include <chrono>



__global__ void preprocess_image(uint8_t* data, int* histo, uint8_t* max_color_array, uint8_t* min_color_array,
                                 uint8_t* pixel_found_array, int* start_indices, int* stop_indices) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    for (int pixel_index=start_indices[idx]; pixel_index<stop_indices[idx]; ++pixel_index) {

        bool alpha_valid = data[pixel_index * 4 + 3] >= 125;
        bool not_white = data[pixel_index * 4] <= 250 || data[pixel_index * 4 + 1] <= 250 || data[pixel_index * 4 + 2] <= 250;
        bool mask = alpha_valid && not_white;

        int histo_pixel_index = 0;

        for (int color_index=0; color_index<3; ++color_index) {
            uint8_t color_value = data[pixel_index * 4 + color_index] >> RSHIFT;

            max_color_array[idx * 3 + color_index] = max(max_color_array[idx * 3 + color_index], color_value * mask);
            min_color_array[idx * 3 + color_index] = min(min_color_array[idx * 3 + color_index], color_value * (1 - mask));

            histo_pixel_index += color_value << ((2 - color_index) * SIGBITS);
        }
        //histo[histo_pixel_index] += 1 * mask;
        atomicAdd(histo + histo_pixel_index, int(mask));
        pixel_found_array[idx] = pixel_found_array[idx] || mask;
    }
}


std::tuple<std::vector<int>, color_t, color_t, bool> get_histo_cuda(uint8_t* data, int pixel_count, int quality) {
    //uint8_t* data = (uint8_t*)image_buffer.ptr;

    std::vector<std::chrono::time_point<std::chrono::system_clock>> times;
    times.push_back(std::chrono::system_clock::now());

    std::vector<int> histo(std::pow(2, 3 * SIGBITS), 0);

    int num_threads = NUM_BLOCKS * THREADS_PER_BLOCK;
    int data_per_thread = std::ceil(double(pixel_count) / double(num_threads));

    std::vector<uint8_t> max_color_array(num_threads * 3, 0);
    std::vector<uint8_t> min_color_array(num_threads * 3, 0);
    std::vector<uint8_t> pixel_found_array(num_threads, 0);
    std::vector<int> start_indices;
    std::vector<int> stop_indices;

    start_indices.reserve(num_threads);
    stop_indices.reserve(num_threads);
    for (int i=0; i<num_threads; ++i) {
      start_indices.push_back(std::min(i * data_per_thread, pixel_count));
      stop_indices.push_back(std::min((i + 1) * data_per_thread, pixel_count));
    }

    if (start_indices.size() != num_threads) {
        std::cout << num_threads << " " << start_indices.size() << std::endl;
        throw std::runtime_error("Bug in preparation of data for cuda");
    }

    times.push_back(std::chrono::system_clock::now());


    uint8_t *cuda_data, *cuda_max_color_array, *cuda_min_color_array;
    int *cuda_histo, *cuda_start_indices, *cuda_stop_indices;
    uint8_t *cuda_pixel_found_array;
    hipMalloc(&cuda_data, pixel_count * 4);
    //hipHostRegister(data, pixel_count * 4, hipHostRegisterReadOnly);
    //hipHostGetDevicePointer((void **) &cuda_data, (void *) data, 0);

    hipMalloc(&cuda_max_color_array, max_color_array.size());
    hipMalloc(&cuda_min_color_array, min_color_array.size());
    hipMalloc(&cuda_histo, histo.size() * sizeof(int));
    hipMalloc(&cuda_start_indices, start_indices.size() * sizeof(int));
    hipMalloc(&cuda_stop_indices, stop_indices.size() * sizeof(int));
    hipMalloc(&cuda_pixel_found_array, pixel_found_array.size() * sizeof(uint8_t));

    hipMemcpy(cuda_data, data, pixel_count * 4, hipMemcpyHostToDevice);
    hipMemcpy(cuda_max_color_array, max_color_array.data(), max_color_array.size(), hipMemcpyHostToDevice);
    hipMemcpy(cuda_min_color_array, min_color_array.data(), min_color_array.size(), hipMemcpyHostToDevice);
    hipMemcpy(cuda_histo, histo.data(), histo.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cuda_start_indices, start_indices.data(), start_indices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cuda_stop_indices, stop_indices.data(), stop_indices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cuda_pixel_found_array, pixel_found_array.data(), pixel_found_array.size() * sizeof(uint8_t), hipMemcpyHostToDevice);

    times.push_back(std::chrono::system_clock::now());

    preprocess_image<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(cuda_data, cuda_histo, cuda_max_color_array, cuda_min_color_array,
                                                        cuda_pixel_found_array, cuda_start_indices, cuda_stop_indices);

    hipMemcpy(histo.data(), cuda_histo, histo.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(max_color_array.data(), cuda_max_color_array, max_color_array.size(), hipMemcpyDeviceToHost);
    hipMemcpy(min_color_array.data(), cuda_min_color_array, min_color_array.size(), hipMemcpyDeviceToHost);
    hipMemcpy(pixel_found_array.data(), cuda_pixel_found_array, pixel_found_array.size() * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipFree(cuda_data);
    hipFree(cuda_max_color_array);
    hipFree(cuda_min_color_array);
    hipFree(cuda_histo);
    hipFree(cuda_start_indices);
    hipFree(cuda_stop_indices);
    hipFree(cuda_pixel_found_array);

    times.push_back(std::chrono::system_clock::now());

    color_t max_color, min_color;
    bool pixel_found = false;
    for (int thread_index=0; thread_index<num_threads; ++thread_index) {
        for (int color_index=0; color_index<3; ++color_index) {
            max_color[color_index] = std::max(max_color[color_index], max_color_array[thread_index * 3 + color_index]);
            min_color[color_index] = std::min(min_color[color_index], min_color_array[thread_index * 3 + color_index]);
        }
        pixel_found = pixel_found_array[thread_index] || pixel_found;
    }

    times.push_back(std::chrono::system_clock::now());

    for (int i=1; i<times.size(); ++i) {
       std::cout << "Times: " << (times[i] - times[i-1]).count() << std::endl;
    }

    return {histo, min_color, max_color, pixel_found};
}
